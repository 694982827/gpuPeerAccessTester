#include <chrono>
#include <iostream>
#include <hip/hip_runtime.h>
#define CUDA_CHECK(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code,
                      const char *file,
                      int line,
                      bool abort = true) {
  if(code != hipSuccess) {
    std::cerr <<"GPUassert: " << hipGetErrorString(code) << " " << file << " "
                              << line << std::endl;
    exit(1);
  }
}


//Copy from other GPUs to this device
double normalMemCpyOther2This(float ** gpu_ptrs, int num_gpus, int bytes_to_transfer_each, int curr_gpu) {
    std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < num_gpus; i++) {
        if (i != curr_gpu) {
            CUDA_CHECK(hipMemcpy(gpu_ptrs[curr_gpu] + (bytes_to_transfer_each/4/num_gpus)*i,
             gpu_ptrs[i] + (bytes_to_transfer_each/4/num_gpus)*i,
             bytes_to_transfer_each/num_gpus,
                hipMemcpyDefault));
        }

    }
    std::chrono::high_resolution_clock::time_point end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> secs = end-start;
    return secs.count();
}

//From this device to other GPUS, async
double normalMemCpyThis2Other(float ** gpu_ptrs, int num_gpus, int bytes_to_transfer_each, int curr_gpu) {
    std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < num_gpus; i++) {
        if (i != curr_gpu) {
            CUDA_CHECK(hipMemcpy(gpu_ptrs[i] + (bytes_to_transfer_each/4/num_gpus)*i,
             gpu_ptrs[curr_gpu] + (bytes_to_transfer_each/4/num_gpus)*i,
             bytes_to_transfer_each/num_gpus,
                hipMemcpyDefault));
        }

    }
    std::chrono::high_resolution_clock::time_point end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> secs = end-start;
    return secs.count();
}

//Copy from other GPUs to this device, async
double normalMemCpyOther2ThisAsync(float ** gpu_ptrs, int num_gpus, int bytes_to_transfer_each, int curr_gpu) {
    hipStream_t streams[num_gpus];
    for (int i = 0; i < num_gpus; i++) {
        CUDA_CHECK(hipStreamCreate(&streams[i]));
    }

    std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < num_gpus; i++) {
        if (i != curr_gpu) {
            CUDA_CHECK(hipMemcpyAsync(gpu_ptrs[curr_gpu] + (bytes_to_transfer_each/4/num_gpus)*i,
             gpu_ptrs[i] + (bytes_to_transfer_each/4/num_gpus)*i,
             bytes_to_transfer_each/num_gpus,
                hipMemcpyDefault, streams[i]));
        }

    for (int i = 0; i < num_gpus; i++) {
        CUDA_CHECK(hipStreamSynchronize(streams[i]));
    }

    }
    std::chrono::high_resolution_clock::time_point end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> secs = end-start;
    return secs.count();
}

//From this device to other GPUS, async
double normalMemCpyThis2OtherAsync(float ** gpu_ptrs, int num_gpus, int bytes_to_transfer_each, int curr_gpu) {
    hipStream_t streams[num_gpus];
    for (int i = 0; i < num_gpus; i++) {
        CUDA_CHECK(hipStreamCreate(&streams[i]));
    }

    std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < num_gpus; i++) {
        if (i != curr_gpu) {
            CUDA_CHECK(hipMemcpyAsync(gpu_ptrs[i] + (bytes_to_transfer_each/4/num_gpus)*i,
             gpu_ptrs[curr_gpu] + (bytes_to_transfer_each/4/num_gpus)*i,
             bytes_to_transfer_each/num_gpus,
                hipMemcpyDefault, streams[i]));
        }

    }

    for (int i = 0; i < num_gpus; i++) {
        CUDA_CHECK(hipStreamSynchronize(streams[i]));
    }

    std::chrono::high_resolution_clock::time_point end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> secs = end-start;
    return secs.count();
}

int main(int argc, char * argv[]) {

    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " num gpus total_model_memory" << std::endl;
        exit(1);
    }

    int num_gpus = atoi(argv[1]);

    int bytes_to_transfer_each = atoi(argv[2])*1024*1024;

    float ** gpu_ptrs = new float*[num_gpus];

    //Initiate with random memory, we don't care what it is. Also init the streams
    for (int i = 0; i < num_gpus; i++) {
        hipSetDevice(i);
        CUDA_CHECK(hipMalloc(&gpu_ptrs[i], bytes_to_transfer_each));
    }

    auto other_this = normalMemCpyOther2This(gpu_ptrs, num_gpus, bytes_to_transfer_each, 0);
    auto this_other = normalMemCpyThis2Other(gpu_ptrs, num_gpus, bytes_to_transfer_each, 0);
    auto other_thisAsync = normalMemCpyOther2ThisAsync(gpu_ptrs, num_gpus, bytes_to_transfer_each, 0);
    auto this_otherAsync = normalMemCpyThis2OtherAsync(gpu_ptrs, num_gpus, bytes_to_transfer_each, 0);
    
    for (int i = 1; i < num_gpus; i++) {
      other_this += normalMemCpyOther2This(gpu_ptrs, num_gpus, bytes_to_transfer_each, i);
      this_other += normalMemCpyThis2Other(gpu_ptrs, num_gpus, bytes_to_transfer_each, i);
      other_thisAsync += normalMemCpyOther2ThisAsync(gpu_ptrs, num_gpus, bytes_to_transfer_each, i);
      this_otherAsync += normalMemCpyThis2OtherAsync(gpu_ptrs, num_gpus, bytes_to_transfer_each, i);
    }

    std::cout << std::fixed << "Other to this took: " << other_this << " seconds." << std::endl;
    std::cout << std::fixed << "This to other took: " << this_other << " seconds." << std::endl;
    std::cout << std::fixed << "Other to this Async took: " << other_this << " seconds." << std::endl;
    std::cout << std::fixed << "This to other Async took: " << this_other << " seconds." << std::endl;

    //Attempt to enable peer access
    for (int i = 0; i<num_gpus; i++) {
        for (int j = 0; j<num_gpus; j++) {
            if (i != j) {
                int result;
                CUDA_CHECK(hipDeviceCanAccessPeer(&result, i, j));
                if (result) {
                    hipSetDevice(i);
                    hipDeviceEnablePeerAccess (j, 0);
                } else {
                    std::cout << std::fixed << "Peer access unavailable between devices: " << i << " and " << j << std::endl;
                }
            }
        }
    }

    //Redo the benchmarks, see if it is any different

    other_this = normalMemCpyOther2This(gpu_ptrs, num_gpus, bytes_to_transfer_each, 0);
    this_other = normalMemCpyThis2Other(gpu_ptrs, num_gpus, bytes_to_transfer_each, 0);
    other_thisAsync = normalMemCpyOther2ThisAsync(gpu_ptrs, num_gpus, bytes_to_transfer_each, 0);
    this_otherAsync = normalMemCpyThis2OtherAsync(gpu_ptrs, num_gpus, bytes_to_transfer_each, 0);
    
    for (int i = 1; i < num_gpus; i++) {
      other_this += normalMemCpyOther2This(gpu_ptrs, num_gpus, bytes_to_transfer_each, i);
      this_other += normalMemCpyThis2Other(gpu_ptrs, num_gpus, bytes_to_transfer_each, i);
      other_thisAsync += normalMemCpyOther2ThisAsync(gpu_ptrs, num_gpus, bytes_to_transfer_each, i);
      this_otherAsync += normalMemCpyThis2OtherAsync(gpu_ptrs, num_gpus, bytes_to_transfer_each, i);
    }

    std::cout << std::fixed << "Peer other to this took: " << other_this << " seconds." << std::endl;
    std::cout << std::fixed << "Peer this to other took: " << this_other << " seconds." << std::endl;
    std::cout << std::fixed << "Peer other to this Async took: " << other_this << " seconds." << std::endl;
    std::cout << std::fixed << "Peer this to other Async took: " << this_other << " seconds." << std::endl;

    return 0;

}
